
#include <hip/hip_runtime.h>
union RGBA32 {
  uint32_t data;
  struct {
    uint8_t r;
	uint8_t g;
    uint8_t b;
    uint8_t a;
  } color;
};

__constant__ float matrixYuvToRgb[3][3];

// -----------------------------------------------------------------------------------------------------------------------------
template<class T>
__device__ inline T clamp(T x, T lower, T upper)
{
  return x < lower ? lower : (x > upper ? upper : x);
}

// -----------------------------------------------------------------------------------------------------------------------------
template<class Rgb, class YuvUnit>
__device__ inline Rgb yuvToRgbPixel(YuvUnit y, YuvUnit u, YuvUnit v)
{
  const int low = 1 << (sizeof(YuvUnit) * 8 - 4);
  const int mid = 1 << (sizeof(YuvUnit) * 8 - 1);
  float fy = (int)y - low;
  float fu = (int)u - mid;
  float fv = (int)v - mid;
  const float maxf = (1 << sizeof(YuvUnit) * 8) - 1.0f;
  YuvUnit r = (YuvUnit)clamp(matrixYuvToRgb[0][0] * fy + matrixYuvToRgb[0][1] * fu + matrixYuvToRgb[0][2] * fv, 0.0f, maxf);
  YuvUnit g = (YuvUnit)clamp(matrixYuvToRgb[1][0] * fy + matrixYuvToRgb[1][1] * fu + matrixYuvToRgb[1][2] * fv, 0.0f, maxf);
  YuvUnit b = (YuvUnit)clamp(matrixYuvToRgb[2][0] * fy + matrixYuvToRgb[2][1] * fu + matrixYuvToRgb[2][2] * fv, 0.0f, maxf);
  Rgb rgb {};
  const int shift = abs((int)sizeof(YuvUnit) - (int)sizeof(rgb.color.r)) * 8;
  if (sizeof(YuvUnit) >= sizeof(rgb.color.r)) {
    rgb.color.r = r >> shift;
    rgb.color.g = g >> shift;
    rgb.color.b = b >> shift;
   }
  else {
    rgb.color.r = r << shift;
    rgb.color.g = g << shift;
    rgb.color.b = b << shift;
  }
  return rgb;
}

// -----------------------------------------------------------------------------------------------------------------------------
extern "C" __global__ void yuvToRgba32(uint8_t* yuv, int yuvPitch, uint8_t* rgb, int rgbPitch, int width, int height)
{
  int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
  int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
  if (x + 1 >= width || y + 1 >= height) return;
  uint8_t* src = yuv + x * sizeof(ushort2) / 2 + y * yuvPitch;
  uint8_t* dst = rgb + x * sizeof(RGBA32)      + y * rgbPitch;
  ushort2 l0 = *(ushort2*)src;
  ushort2 l1 = *(ushort2*)(src + yuvPitch);
  ushort2 ch = *(ushort2*)(src + (height - y / 2) * yuvPitch);
  *(uint2*)dst = uint2 {
    yuvToRgbPixel<RGBA32>(l0.x, ch.x, ch.y).data,
    yuvToRgbPixel<RGBA32>(l0.y, ch.x, ch.y).data
  };
  *(uint2*)(dst + rgbPitch) = uint2 {
    yuvToRgbPixel<RGBA32>(l1.x, ch.x, ch.y).data, 
    yuvToRgbPixel<RGBA32>(l1.y, ch.x, ch.y).data
  };
}
